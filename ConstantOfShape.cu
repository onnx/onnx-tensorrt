#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2018, NVIDIA CORPORATION. All rights reserved.
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.  IN NO EVENT SHALL
 * THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 */

#include <algorithm>
#include <hip/hip_fp16.h>
#include <cassert>

#include "ConstantOfShape.hpp"

//sds:�������split��˵�������index������0����ʾֻ��һ�����
//sds: index,The index of the output tensor.
nvinfer1::Dims ConstantOfShapePlugin::getOutputDimensions(int index,
                                                const nvinfer1::Dims *inputDims,
                                                int nbInputs) {
  // 'equal' and 'less' is 2, 'where' is 3
  assert(nbInputs == 1);
  assert(index == 0);// only one output
  nvinfer1::Dims const& input_dims = inputDims[0];
  //output_dims == input_dims[0]
  nvinfer1::Dims output_dims = input_dims;
  return output_dims;
}

int ConstantOfShapePlugin::initialize() {
  nvinfer1::Dims dims = this->getInputDims(0);
  _numbers = 1;
  for( int i=dims.nbDims-1; i>=0; i-- ) {
    _numbers *= dims.d[i];
  }
  return 0;
}  



template<typename T>
__global__ void constant_shape_kernel(const int n, T value, T const* __restrict__ y) {
    for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < (n); index += blockDim.x * gridDim.x)
    {
        y[index] = value;
    }
  }
}

  
//sds:�����inputs�����Դ棬outputs�����ڴ档
//sds:ÿ��plugin����enqueue����������inputs�п������ڴ�����Դ棬��addPluginV2ʱ�����inputs����
//    inputs��һ��ָ�룬ά����Ϣ��Ҫ�Լ���ʼ����������initialize�л���enqueue�С�
//    outputs? outputs�Ѿ�����getOutputDimsָ���ĳ�ʼ����Ӧ�ö���gpuָ���?
int ConstantOfShapePlugin::enqueue(int batchSize,
                         const void *const *inputs, void **outputs,
                         void *workspace, hipStream_t stream) {

  //float  const* idata1    = reinterpret_cast<float  const*>(inputs[0]);
  float const* odatas = reinterpret_cast<float const*>(outputs[0]);

  dim3 block(512);
  dim3 grid((_numbers + 512 - 1) / 512);
      
  constant_shape_kernel<<<_numbers, grid, block, 0, stream>>>(_value, odatas);

  return hipGetLastError() != hipSuccess;
}
